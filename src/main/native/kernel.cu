
#include <hip/hip_runtime.h>
extern "C"
__global__ void add(int *a, int *b, int *c, int size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < size) {
        c[idx] = a[idx] + b[idx];
    }
}
