
#include <hip/hip_runtime.h>
extern "C"
__global__ void cat(int m, float *x, float *y, int xSize, int ySize, float *output){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < m){
			int side = i % (xSize + ySize);
			int index = i / (xSize + ySize);
			if (side < xSize)
					output[i] = x[index * xSize + side];
			else
					output[i] = y[index * ySize + side - xSize];
	}
}