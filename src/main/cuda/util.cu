
#include <hip/hip_runtime.h>
extern "C"
__global__ void cat(int m, float *x, float *y, int xSize, int ySize, float *output){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < m){
			int side = i % (xSize + ySize);
			int index = i / (xSize + ySize);
			if (side < xSize)
					output[i] = x[index * xSize + side];
			else
					output[i] = y[index * ySize + side - xSize];
	}
}

// m, n - shape of input
extern "C"
__global__ void matrixTransposition(int m, int n, float *input, float *output){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if(i < n && j < m)
		output[i * m + j] = input[j * n + i];
}

extern "C"
__global__ void broadcasting(int elems2copy, int copyCount, int n, float* input, float* output){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i < n){
		int blockIdx = i / elems2copy / copyCount;
		int valueIdx = i % elems2copy;
		output[i] = input[blockIdx * elems2copy + valueIdx];
	}
}


extern "C"
__global__ void indexSelection(int s, int a, int f, int n, float *input, float *output){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n){
		int n = i / s;
		int r = i % s;
		int index = s * (a * n + f) + r;
		
		output[i] = input[index];
	}
}